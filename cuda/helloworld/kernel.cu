#include "hip/hip_runtime.h"
// kernel.cu
#include <vector>
#include "hip/hip_runtime.h"
#include ""
#include "iostream"
#include "kernel.cuh"
// CUDA kernel function
__global__ void myKernel(int *d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = index * index;
    }
}

void useCUDA()
{

    const int size = 100;
    std::vector<int> h_array(size);
    int *d_array;

    // Allocate GPU memory
    hipMalloc(&d_array, size * sizeof(int));

    // Call the CUDA kernel
    myKernel<<<1, size>>>(d_array, size);

    // Copy data back from GPU to CPU
    hipMemcpy(h_array.data(), d_array, size * sizeof(int), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_array);

    // Print results
    for (int i = 0; i < size; ++i) {
        std::cout << h_array[i] << " ";
    }
    std::cout << std::endl;

}