#include "hip/hip_runtime.h"
#include "foo.cuh"
#include "vector"
#include "iostream"

#define CHECK(res) { if(res != hipSuccess){printf("Error ：%s:%d , ", __FILE__,__LINE__);   \
printf("code : %d , reason : %s \n", res,hipGetErrorString(res));exit(-1);}}


__global__ void foo()
{
    printf("CUDA!\n");
}

// CUDA kernel function
__global__ void myKernel(int *d_array, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        d_array[index] = index * index;
    }
}


void useCUDA()
{

    foo<<<1,5>>>();
    CHECK(hipDeviceSynchronize());

    const int size = 100;
    std::vector<int> h_array(size);
    int *d_array;

    // Allocate GPU memory
    hipMalloc(&d_array, size * sizeof(int));

    // Call the CUDA kernel
    myKernel<<<1, size>>>(d_array, size);

    // Copy data back from GPU to CPU
    hipMemcpy(h_array.data(), d_array, size * sizeof(int), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_array);

    // Print results
    for (int i = 0; i < size; ++i) {
        std::cout << h_array[i] << " ";
    }
    std::cout << std::endl;
}
